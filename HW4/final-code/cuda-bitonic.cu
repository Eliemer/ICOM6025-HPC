
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

double get_walltime()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (double) (tp.tv_sec + tp.tv_usec*1e-6);
}

void file_results(double time_elapsed, int k) {
  FILE *fptr;
  fptr = fopen("cuda_times.txt", "a");

  if(fptr == NULL){
    printf("\nError creating or opening file\n");
    exit(1);
  }

  fprintf(fptr, "cuda\npower of 2: %d\ntime elapsed: %f\n\n", k, time_elapsed);
  fclose(fptr);
}

void compare_and_swap(int *arr,int a, int b, int dir){
  if(dir == (arr[a] > arr[b])){
    int temp = arr[a];
    arr[a] = arr[b];
    arr[b] = temp;
  }
}

void generate_vector(int *D, int n){
  for (size_t i = 0; i < n; i++)
    D[i] = rand()%100;
}

void print_array(int *D, int n) {
  for (size_t i = 0; i < n; i++) {
    printf("%d  ", D[i]);
  }
}

__global__
void bitonic_sort_step(int *D, int n, int a, int b){
  int i, ixa, temp;
  i = threadIdx.x + (blockDim.x * blockIdx.x);
  ixa = i^a;

  if (ixa > i){
    if ((i & b) == 0){
      if (D[i] > D[ixa]){
        temp = D[i];
        D[i] = D[ixa];
        D[ixa] = temp;
      }
    }
    if ((i & b) != 0){
      if (D[i] < D[ixa]){
        temp = D[i];
        D[i] = D[ixa];
        D[ixa] = temp;
      }
    }
  }
    //print_array(D, n);
    //printf("\t\ti: %d\tixa: %d\ta: %d\tb: %d\n", i, ixa, a, b);
}

void bitonic_sort(int *D, int n, int k){
  int a, b;
  int thread_num=512;
  int block_num=1;
  int *dev_D;

  if (k > 9){
    block_num = pow(2, k-8);
  }
  while (thread_num > n){
    thread_num >>= 1;
  }

  hipMalloc((void**) &dev_D, sizeof(int) * n);
  hipMemcpy(dev_D, D, sizeof(int) * n, hipMemcpyHostToDevice);

  dim3 blocks(block_num);
  dim3 threads(thread_num);

  for (b = 2; b <= n; b <<= 1){
    for (a = b>>1; a > 0; a >>= 1){
      bitonic_sort_step<<<blocks, threads>>>(dev_D, n, a, b);
    }
  }

  hipMemcpy(D, dev_D, sizeof(int) * n, hipMemcpyDeviceToHost);
  hipFree(dev_D);
}

int main(int argc, char const *argv[]) {
  /* code */
  int *D; // Array of values
  int n, k; // 2^k = n, number of elements in the array
  double time0, time1;

  if (argc > 1){
    k = atoi(argv[1]);
  } else {
    printf("number of nodes = 2^k\nenter the value of k: \n");
    scanf("%d", &k);
  }

  if (argc > 2){
    srand(atoi(argv[2]));
  } else {
    srand(get_walltime());
  }

  printf("\npopulating array\n");
  n = pow(2, k);
  D = (int*)malloc(sizeof(int)*n);
  generate_vector(D, n);

  if (n < 33){
    printf("printing array\n\n");
    for (size_t i = 0; i < n; i++) {
      printf("%d ", D[i]);
    }
  }

  printf("\n\nbegin bitonic sort\n");
  time0 = get_walltime();
  bitonic_sort(D, n, k);
  time1 = get_walltime();

  if (n < 33){
    printf("finalize bitonic sort\n\n");
    for (size_t i = 0; i < n; i++) {
      printf("%d ", D[i]);
    }
  }

  printf("\n\ntime elapsed: %f s\n\n ", time1 - time0);
  file_results(time1-time0, k);

  free(D);
  return 0;
}
